#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <chrono>
#include <thread>
#include "main.h"

u64 time() {
  auto now = std::chrono::system_clock::now().time_since_epoch();
  return std::chrono::duration_cast<std::chrono::milliseconds>(now).count();
}

void printState(const u8* r) {
  printf("%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x\n", r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11], r[12], r[13], r[14], r[15], r[16], r[17], r[18], r[19], r[20], r[21], r[22], r[23], r[24], r[25], r[26], r[27], r[28], r[29], r[30], r[31]);
}

void Forward1(u8 *c) {
  u8 d[ILEN];
  for(u8 j = 0; j < ILEN; j++)
    d[j] = confusion[c[j]];
  memcpy(c, d, ILEN);
}

void Forward2(u8 *d) {
  u8 c[ILEN];
  c[0] = d[0]^d[7]^d[10]^d[12]^d[13]^d[15]^d[18]^d[19]^d[21]^d[22]^d[25]^d[28]^d[29]^d[30]^d[31];
  c[1] = d[1]^d[4]^d[7]^d[10]^d[11]^d[12]^d[14]^d[15]^d[16]^d[18]^d[21]^d[23]^d[25]^d[26]^d[28];
  c[2] = d[2]^d[5]^d[8]^d[11]^d[13]^d[15]^d[16]^d[17]^d[19]^d[20]^d[22]^d[26]^d[27]^d[28]^d[29];
  c[3] = d[3]^d[6]^d[9]^d[12]^d[14]^d[17]^d[18]^d[20]^d[21]^d[23]^d[24]^d[27]^d[28]^d[29]^d[30];
  c[4] = d[3]^d[4]^d[8]^d[9]^d[11]^d[14]^d[17]^d[18]^d[22]^d[23]^d[24]^d[25]^d[26]^d[27]^d[29];
  c[5] = d[0]^d[3]^d[5]^d[8]^d[10]^d[11]^d[14]^d[15]^d[17]^d[19]^d[20]^d[22]^d[24]^d[29]^d[30];
  c[6] = d[1]^d[6]^d[9]^d[11]^d[12]^d[15]^d[16]^d[18]^d[20]^d[21]^d[23]^d[24]^d[25]^d[30]^d[31];
  c[7] = d[2]^d[7]^d[8]^d[10]^d[13]^d[16]^d[17]^d[19]^d[21]^d[22]^d[24]^d[25]^d[26]^d[28]^d[31];
  c[8] = d[2]^d[4]^d[5]^d[7]^d[8]^d[15]^d[17]^d[20]^d[21]^d[22]^d[23]^d[26]^d[27]^d[29]^d[30];
  c[9] = d[2]^d[3]^d[4]^d[6]^d[7]^d[9]^d[12]^d[15]^d[17]^d[18]^d[20]^d[24]^d[26]^d[29]^d[31];
  c[10] = d[0]^d[3]^d[5]^d[7]^d[10]^d[13]^d[18]^d[19]^d[20]^d[21]^d[24]^d[25]^d[27]^d[28]^d[30];
  c[11] = d[1]^d[4]^d[6]^d[11]^d[14]^d[16]^d[19]^d[20]^d[21]^d[22]^d[25]^d[26]^d[28]^d[29]^d[31];
  c[12] = d[0]^d[1]^d[3]^d[6]^d[11]^d[12]^d[16]^d[17]^d[18]^d[19]^d[21]^d[25]^d[26]^d[30]^d[31];
  c[13] = d[0]^d[2]^d[3]^d[6]^d[7]^d[8]^d[11]^d[13]^d[16]^d[21]^d[22]^d[25]^d[27]^d[28]^d[30];
  c[14] = d[1]^d[3]^d[4]^d[7]^d[9]^d[14]^d[16]^d[17]^d[22]^d[23]^d[24]^d[26]^d[28]^d[29]^d[31];
  c[15] = d[0]^d[2]^d[5]^d[10]^d[15]^d[16]^d[17]^d[18]^d[20]^d[23]^d[24]^d[25]^d[27]^d[29]^d[30];
  c[16] = d[2]^d[3]^d[5]^d[6]^d[9]^d[12]^d[13]^d[14]^d[15]^d[16]^d[23]^d[26]^d[28]^d[29]^d[31];
  c[17] = d[0]^d[2]^d[5]^d[7]^d[9]^d[10]^d[12]^d[17]^d[20]^d[23]^d[26]^d[27]^d[28]^d[30]^d[31];
  c[18] = d[0]^d[1]^d[3]^d[4]^d[6]^d[10]^d[11]^d[12]^d[13]^d[18]^d[21]^d[24]^d[27]^d[29]^d[31];
  c[19] = d[1]^d[2]^d[4]^d[5]^d[7]^d[8]^d[11]^d[12]^d[13]^d[14]^d[19]^d[22]^d[25]^d[28]^d[30];
  c[20] = d[1]^d[2]^d[6]^d[7]^d[8]^d[9]^d[10]^d[11]^d[13]^d[19]^d[20]^d[24]^d[25]^d[27]^d[30];
  c[21] = d[1]^d[3]^d[4]^d[6]^d[8]^d[13]^d[14]^d[16]^d[19]^d[21]^d[24]^d[26]^d[27]^d[30]^d[31];
  c[22] = d[0]^d[2]^d[4]^d[5]^d[7]^d[8]^d[9]^d[14]^d[15]^d[17]^d[22]^d[25]^d[27]^d[28]^d[31];
  c[23] = d[0]^d[1]^d[3]^d[5]^d[6]^d[8]^d[9]^d[10]^d[12]^d[15]^d[18]^d[23]^d[24]^d[26]^d[29];
  c[24] = d[1]^d[4]^d[5]^d[6]^d[7]^d[10]^d[11]^d[13]^d[14]^d[18]^d[20]^d[21]^d[23]^d[24]^d[31];
  c[25] = d[1]^d[2]^d[4]^d[8]^d[10]^d[13]^d[15]^d[18]^d[19]^d[20]^d[22]^d[23]^d[25]^d[28]^d[31];
  c[26] = d[2]^d[3]^d[4]^d[5]^d[8]^d[9]^d[11]^d[12]^d[14]^d[16]^d[19]^d[21]^d[23]^d[26]^d[29];
  c[27] = d[0]^d[3]^d[4]^d[5]^d[6]^d[9]^d[10]^d[12]^d[13]^d[15]^d[17]^d[20]^d[22]^d[27]^d[30];
  c[28] = d[0]^d[1]^d[2]^d[3]^d[5]^d[9]^d[10]^d[14]^d[15]^d[16]^d[17]^d[19]^d[22]^d[27]^d[28];
  c[29] = d[0]^d[5]^d[6]^d[9]^d[11]^d[12]^d[14]^d[16]^d[18]^d[19]^d[22]^d[23]^d[24]^d[27]^d[29];
  c[30] = d[0]^d[1]^d[6]^d[7]^d[8]^d[10]^d[12]^d[13]^d[15]^d[17]^d[19]^d[20]^d[23]^d[25]^d[30];
  c[31] = d[0]^d[1]^d[2]^d[4]^d[7]^d[8]^d[9]^d[11]^d[13]^d[14]^d[16]^d[18]^d[21]^d[26]^d[31];
  memcpy(d, c, ILEN);
}

bool Forward3(const u8 *c) {
  for(u8 i = 0; i < OLEN; i++) {
    u8 fw = confusion[c[i * 2]] ^ confusion[c[i * 2 + 1] + 256];
    if(fw != target[i])
      return false;
  }
  return true;
}

void Forward(u32 index) {
  u8 c[ILEN];
  u64 start = index * BATCH_SIZE;
  u64 end = start + BATCH_SIZE;

  for(u64 batch = start; batch < end; batch++) {
    memcpy(c, &batch, sizeof(u64));
    for(u32 i = 0; i < ROUNDS; i++) {
      Forward1(c);
      Forward2(c);
    }
    if(Forward3(c))
      printf("%ld\n", batch);
  }
}

void Reverse3(uint64_t a, u8* out) {
  for(int i=0;i<OLEN; i++) {
    auto tr = rev3[i][a & 0xFF];
    out[i*2] = tr[0];
    out[i*2+1] = tr[1];
    a = a >> 8;
  }
}

void Reverse1(state& state);
void Reverse21(state state);

void Reverse1Fork(state& state1, u16 extra) {
  state state2;
  u8 data2[ILEN];
  memcpy(data2, state1.data, ILEN);
  data2[state1.pos] = extra;
  state2.data = data2;
  state2.pos = state1.pos+1;
  state2.round = state1.round;
  Reverse1(state2);
}

void Reverse1(state& state) {
  if(state.pos==0) // only do this check once, not for recursive calls
    for (u8 pos_copy = 0; pos_copy < ILEN; pos_copy++) {
      if (!rev1[state.data[pos_copy]]) {
        return;
      }
    }

  for(;state.pos < ILEN; state.pos++) {
    u16 tmp = rev1[state.data[state.pos]]-1;
    if(tmp > 256)
      Reverse1Fork(state, tmp >> 8);
    state.data[state.pos] = tmp&0xFF;
  }
  state.round++;
  state.pos = 0;
  Reverse21(state);
}

void Reverse21(state state) {
  if(state.round < ROUNDS) {
    Forward2(state.data);
    Reverse1(state);
  } else
    printState(state.data);
}

void Reverse(u32 index) {
  u64 start = index * BATCH_SIZE;
  u64 end = start + BATCH_SIZE;

  for (u64 batch = start; batch < end; batch++) {
    u8 r[ILEN];
    state s;
    s.data=r;
    Reverse3(batch, r);
    Reverse21(s);
  }
}


int main() {
  u64 start = time();
  std::thread threads[NR_CORES];
  for(int i=0;i<NR_CORES;i++)
    threads[i] = std::thread(Reverse, i);
  for(int i=0;i<NR_CORES;i++)
    threads[i].join();
  printf("Execution time = %ld\n", time()-start);
}
